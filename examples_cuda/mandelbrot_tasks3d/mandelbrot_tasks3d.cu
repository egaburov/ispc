#include "hip/hip_runtime.h"
/*
  Copyright (c) 2010-2012, Intel Corporation
  All rights reserved.

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are
  met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.

    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

    * Neither the name of Intel Corporation nor the names of its
      contributors may be used to endorse or promote products derived from
      this software without specific prior written permission.


   THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
   IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
   TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
   PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER
   OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
   EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
   PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
   PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
   LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
   NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
   SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.  
*/

#define programCount 32
#define programIndex (threadIdx.x & 31)
#define taskIndex0 (blockIdx.x*4 + (threadIdx.x >> 5))
#define taskCount0 (gridDim.x*4)
#define taskIndex1 (blockIdx.y)
#define taskCount1 (gridDim.y)
#define warpIdx (threadIdx.x >> 5)


__device__
static inline int
mandel(float c_re, float c_im, int count) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i) {
        if (z_re * z_re + z_im * z_im > 4.0f)
            break;

        float new_re = z_re*z_re - z_im*z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}


/* Task to compute the Mandelbrot iterations for a single scanline.
 */
__global__ void
mandelbrot_scanline( float x0,  float dx, 
                     float y0,  float dy,
                     int width,  int height, 
                     int xspan,  int yspan,
                     int maxIterations,  int output[]) {
    const  int xstart = taskIndex0 * xspan;
    const  int xend   = min(xstart  + xspan, width);

    const  int ystart = taskIndex1 * yspan;
    const  int yend   = min(ystart  + yspan, height);
   
  for ( int yi = ystart; yi < yend; yi++)
    for ( int xi = xstart; xi < xend; xi += programCount)
    {
      const float x = x0 + (xi + programIndex) * dx;
      const float y = y0 +  yi              * dy;

      const int res = mandel(x,y,maxIterations);
      const int index = yi * width + (xi + programIndex);
      if (xi + programIndex < xend)
        output[index] = res;
    }
}

__global__ void
mandelbrot_ispc_dev( float x0,  float y0, 
                 float x1,  float y1,
                 int width,  int height, 
                 int maxIterations,  int output[]) {
     float dx = (x1 - x0) / width;
     float dy = (y1 - y0) / height;
    const  int xspan = 64;  /* make sure it is big enough to avoid false-sharing */
    const  int yspan = 8; 


    if (programIndex == 0)
      mandelbrot_scanline<<<dim3((width+4-1)/xspan/4,height/yspan),128>>>
        (x0, dx, y0, dy, width, height, xspan, yspan,  maxIterations, output);
    hipDeviceSynchronize();
}

extern "C"
void mandelbrot_ispc(float x0, float y0, 
    float x1, float y1,
    int width, int height, 
    int maxIterations, int output[])
{
  mandelbrot_ispc_dev<<<1,32>>>
    (x0,y0,x1,y1,width,height,maxIterations,output);
  hipDeviceSynchronize();
}

