#include "hip/hip_runtime.h"
#define programCount 32
#define programIndex (threadIdx.x & 31)
#define taskIndex0 (blockIdx.x*4 + (threadIdx.x >> 5))
#define taskIndex1 (blockIdx.y)
#define taskIndex2 (blockIdx.z)
#define taskCount0  (gridDim.x*4)
#define taskCount1  (gridDim.y)
#define taskCount2  (gridDim.z)

__device__ static void
stencil_step( int x0,  int x1,
              int y0,  int y1,
              int z0,  int z1,
              int Nx,  int Ny,  int Nz,
              const double coef[4],  const double vsq[],
              const double Ain[],  double Aout[]) {
    const  int Nxy = Nx * Ny;


#if 0
    foreach (z = z0 ... z1, y = y0 ... y1, x = x0 ... x1) {
#else
      const  double coef0 = coef[0];
      const  double coef1 = coef[1];
      const  double coef2 = coef[2];
      const  double coef3 = coef[3];
      for ( int z = z0; z < z1; z++)
        for ( int y = y0 ; y < y1; y++)
          for ( int xb = x0; xb < x1; xb += programCount)
          {
            const int x = xb + programIndex;

#endif
            int index = (z * Nxy) + (y * Nx) + x;
#define A_cur(x, y, z) __ldg(&Ain[index + (x) + ((y) * Nx) + ((z) * Nxy)])
#define A_next(x, y, z) Aout[index + (x) + ((y) * Nx) + ((z) * Nxy)]
            double div = 
              coef0 *  A_cur(0, 0, 0) +
              coef1 * (A_cur(+1, 0, 0) + A_cur(-1, 0, 0) +
                  A_cur(0, +1, 0) + A_cur(0, -1, 0) +
                  A_cur(0, 0, +1) + A_cur(0, 0, -1)) +
              coef2 * (A_cur(+2, 0, 0) + A_cur(-2, 0, 0) +
                  A_cur(0, +2, 0) + A_cur(0, -2, 0) +
                  A_cur(0, 0, +2) + A_cur(0, 0, -2)) +
              coef3 * (A_cur(+3, 0, 0) + A_cur(-3, 0, 0) +
                  A_cur(0, +3, 0) + A_cur(0, -3, 0) +
                  A_cur(0, 0, +3) + A_cur(0, 0, -3));

            if (x < x1)
              A_next(0, 0, 0) = 2.0 * A_cur(0, 0, 0) - A_next(0, 0, 0) + 
                __ldg(&vsq[index]) * div;
          }
}


#define SPANX 32
#define SPANY 2
#define SPANZ 4

__global__  void
stencil_step_task( int x0,  int x1,
                   int y0,  int y1,
                   int z0,  int z1,
                   int Nx,  int Ny,  int Nz,
                   const double coef[4],  const double vsq[],
                   const double Ain[],  double Aout[]) {
  if (taskIndex0 >= taskCount0 || 
      taskIndex1 >= taskCount1 || 
      taskIndex2 >= taskCount2)
    return;

  const  int xfirst = x0 + taskIndex0 * SPANX;
  const  int xlast  = min(x1, xfirst + SPANX);

  const  int yfirst = y0 + taskIndex1 * SPANY;
  const  int ylast  = min(y1, yfirst + SPANY);

  const  int zfirst = z0 + taskIndex2 * SPANZ;
  const  int zlast  = min(z1, zfirst + SPANZ);

  stencil_step(xfirst,xlast, yfirst,ylast, zfirst,zlast,
      Nx, Ny, Nz, coef, vsq, Ain, Aout);
}



extern "C"
__global__ void
loop_stencil_ispc_tasks( int t0,  int t1, 
                         int x0,  int x1,
                         int y0,  int y1,
                         int z0,  int z1,
                         int Nx,  int Ny,  int Nz,
                         const double coef[4], 
                         const double vsq[],
                         double Aeven[],  double Aodd[])
{
#define NB(x,n) (((x)+(n)-1)/(n))

  dim3 grid((NB(x1-x0,SPANX)-1)/4+1, NB(y1-y0,SPANY), NB(z1-z0,SPANZ));

    for ( int t = t0; t < t1; ++t) 
    {
      // Parallelize across cores as well: each task will work on a slice
      // of 1 in the z extent of the volume.
      if ((t & 1) == 0)
      {
        if (programIndex == 0)
          stencil_step_task<<<grid,128>>>(x0, x1, y0, y1, z0, z1, Nx, Ny, Nz, 
              coef, vsq, Aeven, Aodd);
      }
      else
      {
        if (programIndex == 0)
          stencil_step_task<<<grid,128>>>(x0, x1, y0, y1, z0, z1, Nx, Ny, Nz, 
              coef, vsq, Aodd, Aeven);
      }

      // We need to wait for all of the launched tasks to finish before
      // starting the next iteration
      hipDeviceSynchronize();
    }
}
