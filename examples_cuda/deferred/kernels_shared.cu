#include "hip/hip_runtime.h"
/*
  Copyright (c) 2010-2011, Intel Corporation
  All rights reserved.

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are
  met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.

    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

    * Neither the name of Intel Corporation nor the names of its
      contributors may be used to endorse or promote products derived from
      this software without specific prior written permission.


   THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
   IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
   TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
   PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER
   OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
   EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
   PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
   PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
   LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
   NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
   SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.  
*/


#include "deferred.h"
#include <stdio.h>
#include <assert.h>

#define programCount 32
#define programIndex (threadIdx.x & 31)
#define taskIndex (blockIdx.x*4 + (threadIdx.x >> 5))
#define taskCount (gridDim.x*4)
#define warpIdx (threadIdx.x >> 5)

#define int32 int
#define int16 short
#define int8 char

__device__ static inline float clamp(float v, float low, float high) 
{
      return min(max(v, low), high);
}

struct InputDataArrays
{
    float *zBuffer;
    unsigned int16 *normalEncoded_x; // half float
    unsigned int16 *normalEncoded_y; // half float
    unsigned int16 *specularAmount; // half float
    unsigned int16 *specularPower; // half float
    unsigned int8 *albedo_x; // unorm8
    unsigned int8 *albedo_y; // unorm8
    unsigned int8 *albedo_z; // unorm8
    float *lightPositionView_x;
    float *lightPositionView_y;
    float *lightPositionView_z;
    float *lightAttenuationBegin;
    float *lightColor_x;
    float *lightColor_y;
    float *lightColor_z;
    float *lightAttenuationEnd;
};

struct InputHeader
{
    float cameraProj[4][4];
    float cameraNear;
    float cameraFar;

    int32 framebufferWidth;
    int32 framebufferHeight;
    int32 numLights;
    int32 inputDataChunkSize;
    int32 inputDataArrayOffsets[idaNum];
};


///////////////////////////////////////////////////////////////////////////
// Common utility routines

__device__
static inline float
dot3(float x, float y, float z, float a, float b, float c) {
    return (x*a + y*b + z*c);
}


#if 0
template<typename T, int N>
struct Uniform
{
  static __shared__ T shdata[128];
  T data[(N-1)/programCount+1];

  __device__ inline const T get(const int i) const
  {
    const int  elemIdx = i & (programCount-1);
    const int chunkIdx = i >> 5;
    return __shfl(data[chunkIdx], elemIdx);
  }
  
  __device__ inline void set(const int i, const T value) const
  {
    const int  elemIdx = i & (programCount-1);
    const int chunkIdx = i >> 5;
    shdata[elemIdx] = value;
    data[chunkIdx]  = shdata[programIndex];
  }
}
#endif


__device__
static inline void
normalize3(float x, float y, float z, float &ox, float &oy, float &oz) {
    float n = rsqrt(x*x + y*y + z*z);
    ox = x * n;
    oy = y * n;
    oz = z * n;
}

__device__ inline
static float reduce_min(float value)
{
#pragma unroll
  for (int i = 4; i >=0; i--)
    value = min(value, __shfl_xor(value, 1<<i, 32));
  return value;
}
__device__ inline
static float reduce_max(float value)
{
#pragma unroll
  for (int i = 4; i >=0; i--)
    value = max(value, __shfl_xor(value, 1<<i, 32));
  return value;
}
__device__ inline
static int reduce_sum(int value)
{
#pragma unroll
  for (int i = 4; i >=0; i--)
    value +=  __shfl_xor(value, 1<<i, 32);
  return value;
}
static __device__ __forceinline__ uint shfl_scan_add_step(uint partial, uint up_offset)
{
  uint result;
  asm(
      "{.reg .u32 r0;"
      ".reg .pred p;"
      "shfl.up.b32 r0|p, %1, %2, 0;"
      "@p add.u32 r0, r0, %3;"
      "mov.u32 %0, r0;}"
      : "=r"(result) : "r"(partial), "r"(up_offset), "r"(partial));
  return result;
}
static __device__ __forceinline__ int inclusive_scan_warp(const int value)
{
  uint sum = value;
#pragma unroll
  for(int i = 0; i < 5; ++i)
    sum = shfl_scan_add_step(sum, 1 << i);
  return sum - value;
}


static __device__ __forceinline__ int lanemask_lt()
{
  int mask;
  asm("mov.u32 %0, %lanemask_lt;" : "=r" (mask));
  return mask;
}
static __device__ __forceinline__ int2 warpBinExclusiveScan(const bool p)
{
  const unsigned int b = __ballot(p);
  return make_int2(__popc(b & lanemask_lt()), __popc(b));
}





__device__
static inline float
Unorm8ToFloat32(unsigned int8 u) {
    return (float)u * (1.0f / 255.0f);
}


__device__
static inline unsigned int8
Float32ToUnorm8(float f) {
    return (unsigned int8)(f * 255.0f);
}


__device__
static inline void
ComputeZBounds(
     int32 tileStartX,  int32 tileEndX,
     int32 tileStartY,  int32 tileEndY,
    // G-buffer data
     float zBuffer[],
     int32 gBufferWidth,
    // Camera data
     float cameraProj_33,  float cameraProj_43,
     float cameraNear,  float cameraFar,
    // Output
     float &minZ,
     float &maxZ
    )
{
    // Find Z bounds
    float laneMinZ = cameraFar;
    float laneMaxZ = cameraNear;
    for ( int32 y = tileStartY; y < tileEndY; ++y) {
        for ( int xb = tileStartX; xb < tileEndX; xb += programCount)
        {
          const int x = xb + programIndex;
          if (x >= tileEndX) break;
            // Unproject depth buffer Z value into view space
            float z = zBuffer[y * gBufferWidth + x];
            float viewSpaceZ = cameraProj_43 / (z - cameraProj_33);

            // Work out Z bounds for our samples
            // Avoid considering skybox/background or otherwise invalid pixels
            if ((viewSpaceZ < cameraFar) && (viewSpaceZ >= cameraNear)) {
                laneMinZ = min(laneMinZ, viewSpaceZ);
                laneMaxZ = max(laneMaxZ, viewSpaceZ);
            }
        }
    }
    minZ = reduce_min(laneMinZ);
    maxZ = reduce_max(laneMaxZ);
}


__device__
static inline  int32
IntersectLightsWithTileMinMax(
     int32 tileStartX,  int32 tileEndX,
     int32 tileStartY,  int32 tileEndY,
    // Tile data
     float minZ,
     float maxZ,
    // G-buffer data
     int32 gBufferWidth,  int32 gBufferHeight,
    // Camera data
     float cameraProj_11,  float cameraProj_22,
    // Light Data
     int32 numLights,
     float light_positionView_x_array[],
     float light_positionView_y_array[],
     float light_positionView_z_array[],
     float light_attenuationEnd_array[],
    // Output
     int32 tileLightIndices[]
    )
{
     float gBufferScale_x = 0.5f * (float)gBufferWidth;
     float gBufferScale_y = 0.5f * (float)gBufferHeight;
        
     float frustumPlanes_xy[4] = {
        -(cameraProj_11 * gBufferScale_x),
         (cameraProj_11 * gBufferScale_x),
         (cameraProj_22 * gBufferScale_y),
        -(cameraProj_22 * gBufferScale_y) };
     float frustumPlanes_z[4] = {
         tileEndX - gBufferScale_x,
        -tileStartX + gBufferScale_x,
         tileEndY - gBufferScale_y,
        -tileStartY + gBufferScale_y };

    for ( int i = 0; i < 4; ++i) {
         float norm = rsqrt(frustumPlanes_xy[i] * frustumPlanes_xy[i] + 
                                   frustumPlanes_z[i] * frustumPlanes_z[i]);
        frustumPlanes_xy[i] *= norm;
        frustumPlanes_z[i] *= norm;
    }

     int32 tileNumLights = 0;

    for ( int lightIndexB = 0; lightIndexB < numLights; lightIndexB += programCount)
    {
      const int lightIndex = lightIndexB + programIndex;

        float light_positionView_z = light_positionView_z_array[lightIndex];
        float light_attenuationEnd = light_attenuationEnd_array[lightIndex];
        float light_attenuationEndNeg = -light_attenuationEnd;

        float d = light_positionView_z - minZ;
        bool inFrustum = (d >= light_attenuationEndNeg);

        d = maxZ - light_positionView_z;
        inFrustum = inFrustum && (d >= light_attenuationEndNeg);
        
        // This seems better than cif(!inFrustum) ccontinue; here since we
        // don't actually need to mask the rest of this function - this is
        // just a greedy early-out.  Could also structure all of this as
        // nested if() statements, but this a bit easier to read
        int active = 0;
        if ((inFrustum)) {
            float light_positionView_x = light_positionView_x_array[lightIndex];
            float light_positionView_y = light_positionView_y_array[lightIndex];

            d = light_positionView_z * frustumPlanes_z[0] + 
                light_positionView_x * frustumPlanes_xy[0];
            inFrustum = inFrustum && (d >= light_attenuationEndNeg);

            d = light_positionView_z * frustumPlanes_z[1] + 
                light_positionView_x * frustumPlanes_xy[1];
            inFrustum = inFrustum && (d >= light_attenuationEndNeg);

            d = light_positionView_z * frustumPlanes_z[2] + 
                light_positionView_y * frustumPlanes_xy[2];
            inFrustum = inFrustum && (d >= light_attenuationEndNeg);

            d = light_positionView_z * frustumPlanes_z[3] + 
                light_positionView_y * frustumPlanes_xy[3];
            inFrustum = inFrustum && (d >= light_attenuationEndNeg);
        
            // Pack and store intersecting lights
#if 0
            if (inFrustum) {
                tileNumLights += packed_store_active(&tileLightIndices[tileNumLights], 
                                                     lightIndex);
            }
#else
            if (inFrustum)
            {
              active = 1;
            }
#endif
        }
#if 1
        if (lightIndex >= numLights) 
          active = 0;

#if 0
        const int idx = tileNumLights + inclusive_scan_warp(active);
        const int nactive = reduce_sum(active);
#else
        const int2 res = warpBinExclusiveScan(active);
        const int idx = tileNumLights + res.x;
        const int nactive = res.y;
#endif
        if (active)
          tileLightIndices[idx] = lightIndex;
        tileNumLights += nactive;
#endif
    }

    return tileNumLights;
}


__device__
static inline   int32
IntersectLightsWithTile(
     int32 tileStartX,  int32 tileEndX,
     int32 tileStartY,  int32 tileEndY,
     int32 gBufferWidth,  int32 gBufferHeight,
    // G-buffer data
     float zBuffer[],
    // Camera data
     float cameraProj_11,  float cameraProj_22,
     float cameraProj_33,  float cameraProj_43,
     float cameraNear,  float cameraFar,
    // Light Data
     int32 numLights,
     float light_positionView_x_array[],
     float light_positionView_y_array[],
     float light_positionView_z_array[],
     float light_attenuationEnd_array[],
    // Output
     int32 tileLightIndices[]
    )
{
     float minZ, maxZ;
    ComputeZBounds(tileStartX, tileEndX, tileStartY, tileEndY,
        zBuffer, gBufferWidth, cameraProj_33, cameraProj_43, cameraNear, cameraFar,
        minZ, maxZ);


     int32 tileNumLights = IntersectLightsWithTileMinMax(
        tileStartX, tileEndX, tileStartY, tileEndY, minZ, maxZ,
        gBufferWidth, gBufferHeight, cameraProj_11, cameraProj_22,
        MAX_LIGHTS, light_positionView_x_array, light_positionView_y_array, 
        light_positionView_z_array, light_attenuationEnd_array,
        tileLightIndices);

    return tileNumLights;
}


__device__
static inline void
ShadeTile(
     int32 tileStartX,  int32 tileEndX,
     int32 tileStartY,  int32 tileEndY,
     int32 gBufferWidth,  int32 gBufferHeight,
    const  InputDataArrays &inputData,
    // Camera data
     float cameraProj_11,  float cameraProj_22,
     float cameraProj_33,  float cameraProj_43,
    // Light list
     int32 tileLightIndices[],
     int32 tileNumLights,
    // UI
     bool visualizeLightCount,
    // Output
     unsigned int8 framebuffer_r[],
     unsigned int8 framebuffer_g[],
     unsigned int8 framebuffer_b[]
    )
{
    if (tileNumLights == 0 || visualizeLightCount) {
         unsigned int8 c = (unsigned int8)(min(tileNumLights << 2, 255));
        for ( int32 y = tileStartY; y < tileEndY; ++y) {
            for ( int xb = tileStartX ; xb < tileEndX; xb += programCount)
            { 
              const int x = xb + programIndex;
              if (x >= tileEndX) continue;
                int32 framebufferIndex = (y * gBufferWidth + x);
                framebuffer_r[framebufferIndex] = c;
                framebuffer_g[framebufferIndex] = c;
                framebuffer_b[framebufferIndex] = c;
            }
        }
    } else {
         float twoOverGBufferWidth = 2.0f / gBufferWidth;
         float twoOverGBufferHeight = 2.0f / gBufferHeight;
        
        for ( int32 y = tileStartY; y < tileEndY; ++y) {
             float positionScreen_y = -(((0.5f + y) * twoOverGBufferHeight) - 1.f);

            for ( int xb = tileStartX ; xb < tileEndX; xb += programCount)
            { 
              const int x = xb + programIndex;
//              if (x >= tileEndX) break;
                int32 gBufferOffset = y * gBufferWidth + x;
                
                // Reconstruct position and (negative) view vector from G-buffer
                float surface_positionView_x, surface_positionView_y, surface_positionView_z;
                float Vneg_x, Vneg_y, Vneg_z;

                float z = inputData.zBuffer[gBufferOffset];

                // Compute screen/clip-space position
                // NOTE: Mind DX11 viewport transform and pixel center!
                float positionScreen_x = (0.5f + (float)(x)) * 
                    twoOverGBufferWidth - 1.0f;

                // Unproject depth buffer Z value into view space
                surface_positionView_z = cameraProj_43 / (z - cameraProj_33);
                surface_positionView_x = positionScreen_x * surface_positionView_z / 
                    cameraProj_11;
                surface_positionView_y = positionScreen_y * surface_positionView_z / 
                    cameraProj_22;
                
                // We actually end up with a vector pointing *at* the
                // surface (i.e. the negative view vector)
                normalize3(surface_positionView_x, surface_positionView_y, 
                           surface_positionView_z, Vneg_x, Vneg_y, Vneg_z);

                // Reconstruct normal from G-buffer
                float surface_normal_x, surface_normal_y, surface_normal_z;
                float normal_x = __half2float(inputData.normalEncoded_x[gBufferOffset]);
                float normal_y = __half2float(inputData.normalEncoded_y[gBufferOffset]);
                    
                float f = (normal_x - normal_x * normal_x) + (normal_y - normal_y * normal_y);
                float m = sqrt(4.0f * f - 1.0f);
                    
                surface_normal_x = m * (4.0f * normal_x - 2.0f);
                surface_normal_y = m * (4.0f * normal_y - 2.0f);
                surface_normal_z = 3.0f - 8.0f * f;

                // Load other G-buffer parameters
                float surface_specularAmount = 
                    __half2float(inputData.specularAmount[gBufferOffset]);
                float surface_specularPower  = 
                    __half2float(inputData.specularPower[gBufferOffset]);
                float surface_albedo_x = Unorm8ToFloat32(inputData.albedo_x[gBufferOffset]);
                float surface_albedo_y = Unorm8ToFloat32(inputData.albedo_y[gBufferOffset]);
                float surface_albedo_z = Unorm8ToFloat32(inputData.albedo_z[gBufferOffset]);
                
                float lit_x = 0.0f;
                float lit_y = 0.0f;
                float lit_z = 0.0f;
                for ( int32 tileLightIndex = 0; tileLightIndex < tileNumLights; 
                     ++tileLightIndex) {
                     int32 lightIndex = tileLightIndices[tileLightIndex];
                                        
                    // Gather light data relevant to initial culling
                     float light_positionView_x = 
                        inputData.lightPositionView_x[lightIndex];
                     float light_positionView_y = 
                        inputData.lightPositionView_y[lightIndex];
                     float light_positionView_z = 
                        inputData.lightPositionView_z[lightIndex];
                     float light_attenuationEnd = 
                        inputData.lightAttenuationEnd[lightIndex];
                    
                    // Compute light vector
                    float L_x = light_positionView_x - surface_positionView_x;
                    float L_y = light_positionView_y - surface_positionView_y;
                    float L_z = light_positionView_z - surface_positionView_z;

                    float distanceToLight2 = dot3(L_x, L_y, L_z, L_x, L_y, L_z);
                    
                    // Clip at end of attenuation
                    float light_attenutaionEnd2 = light_attenuationEnd * light_attenuationEnd;

                    if (distanceToLight2 < light_attenutaionEnd2) {                    
                        float distanceToLight = sqrt(distanceToLight2);

                        // HLSL "rcp" is allowed to be fairly inaccurate
                        float distanceToLightRcp = 1.0f/distanceToLight;
                        L_x *= distanceToLightRcp;
                        L_y *= distanceToLightRcp;
                        L_z *= distanceToLightRcp;

                        // Start computing brdf
                        float NdotL = dot3(surface_normal_x, surface_normal_y, 
                                           surface_normal_z, L_x, L_y, L_z);
                    
                        // Clip back facing
                        if (NdotL > 0.0f) {
                             float light_attenuationBegin = 
                                inputData.lightAttenuationBegin[lightIndex];

                            // Light distance attenuation (linstep)
                            float lightRange = (light_attenuationEnd - light_attenuationBegin);
                            float falloffPosition = (light_attenuationEnd - distanceToLight);
                            float attenuation = min(falloffPosition / lightRange, 1.0f);

                            float H_x = (L_x - Vneg_x);
                            float H_y = (L_y - Vneg_y);
                            float H_z = (L_z - Vneg_z);
                            normalize3(H_x, H_y, H_z, H_x, H_y, H_z);
                    
                            float NdotH = dot3(surface_normal_x, surface_normal_y, 
                                               surface_normal_z, H_x, H_y, H_z);
                            NdotH = max(NdotH, 0.0f);

                            float specular = pow(NdotH, surface_specularPower);
                            float specularNorm = (surface_specularPower + 2.0f) * 
                                (1.0f / 8.0f);
                            float specularContrib = surface_specularAmount * 
                                specularNorm * specular;

                            float k = attenuation * NdotL * (1.0f + specularContrib);
                    
                             float light_color_x = inputData.lightColor_x[lightIndex];
                             float light_color_y = inputData.lightColor_y[lightIndex];
                             float light_color_z = inputData.lightColor_z[lightIndex];

                            float lightContrib_x = surface_albedo_x * light_color_x;
                            float lightContrib_y = surface_albedo_y * light_color_y;
                            float lightContrib_z = surface_albedo_z * light_color_z;

                            lit_x += lightContrib_x * k;
                            lit_y += lightContrib_y * k;
                            lit_z += lightContrib_z * k;
                        }
                    }
                }

                // Gamma correct
                // These pows are pretty slow right now, but we can do
                // something faster if really necessary to squeeze every
                // last bit of performance out of it
                float gamma = 1.0 / 2.2f;
                lit_x = pow(clamp(lit_x, 0.0f, 1.0f), gamma);
                lit_y = pow(clamp(lit_y, 0.0f, 1.0f), gamma);
                lit_z = pow(clamp(lit_z, 0.0f, 1.0f), gamma);
                
                framebuffer_r[gBufferOffset] = Float32ToUnorm8(lit_x);
                framebuffer_g[gBufferOffset] = Float32ToUnorm8(lit_y);
                framebuffer_b[gBufferOffset] = Float32ToUnorm8(lit_z);
            }
        }
    }
}


///////////////////////////////////////////////////////////////////////////
// Static decomposition

extern "C" __global__ void
RenderTile( int num_groups_x,  int num_groups_y,
           const  InputHeader *inputHeaderPtr,
           const  InputDataArrays *inputDataPtr,
            int visualizeLightCount,
           // Output
            unsigned int8 framebuffer_r[],
            unsigned int8 framebuffer_g[],
            unsigned int8 framebuffer_b[]) {
  if (taskIndex >= taskCount) return;

  const  InputHeader inputHeader = *inputHeaderPtr;
  const  InputDataArrays inputData = *inputDataPtr;
     int32 group_y = taskIndex / num_groups_x;
     int32 group_x = taskIndex % num_groups_x;

     int32 tile_start_x = group_x * MIN_TILE_WIDTH;
     int32 tile_start_y = group_y * MIN_TILE_HEIGHT;
     int32 tile_end_x = tile_start_x + MIN_TILE_WIDTH;
     int32 tile_end_y = tile_start_y + MIN_TILE_HEIGHT;

     int framebufferWidth = inputHeader.framebufferWidth;
     int framebufferHeight = inputHeader.framebufferHeight;
     float cameraProj_00 = inputHeader.cameraProj[0][0];
     float cameraProj_11 = inputHeader.cameraProj[1][1];
     float cameraProj_22 = inputHeader.cameraProj[2][2];
     float cameraProj_32 = inputHeader.cameraProj[3][2];

    // Light intersection: figure out which lights illuminate this tile.
#if 0
     int tileLightIndices[MAX_LIGHTS];  // Light list for the tile
#else
     __shared__ int tileLightIndicesFull[4*MAX_LIGHTS];  // Light list for the tile
     int *tileLightIndices = &tileLightIndicesFull[warpIdx*MAX_LIGHTS];
#endif
     int numTileLights = 
        IntersectLightsWithTile(tile_start_x, tile_end_x, 
                                tile_start_y, tile_end_y,
                                framebufferWidth, framebufferHeight,
                                inputData.zBuffer,
                                cameraProj_00, cameraProj_11,
                                cameraProj_22, cameraProj_32,
                                inputHeader.cameraNear, inputHeader.cameraFar,
                                MAX_LIGHTS,
                                inputData.lightPositionView_x, 
                                inputData.lightPositionView_y, 
                                inputData.lightPositionView_z, 
                                inputData.lightAttenuationEnd,
                                tileLightIndices);

    // And now shade the tile, using the lights in tileLightIndices
    ShadeTile(tile_start_x, tile_end_x, tile_start_y, tile_end_y,
              framebufferWidth, framebufferHeight, inputData,
              cameraProj_00, cameraProj_11, cameraProj_22, cameraProj_32,
              tileLightIndices, numTileLights, visualizeLightCount, 
              framebuffer_r, framebuffer_g, framebuffer_b);
}


